#include "hip/hip_runtime.h"
//
// ServerKernels.cu
//
//

#include "Profile.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h> 
#include <hiprand.h>
#include <iostream>
#include <assert.h>
#include <vector>

#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>

// ==================================================================
//  CudaEventTimer
//
// Example usage:
//
//	T.Begin();
//	generate << < blocks, threads >> > (gData, DataPerBlock);
//	T.End();
//	hipError_t err = hipGetLastError();
//	printf("\nError = %s", hipGetErrorString(err));
//	printf("\nDuration of generate kernel = %.3f ms for %d floats\n\n", T.GetTime(), N);
// ==================================================================
class CudaEventTimer
{
public:
	CudaEventTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}
	~CudaEventTimer() { hipEventDestroy(start); hipEventDestroy(stop); }

	void Begin() { hipEventRecord(start); }
	void End() {
		hipEventRecord(stop);
		hipEventSynchronize(stop);
	}
	float GetTime(void) { hipEventElapsedTime(&ms, start, stop); return ms; }

private:
	hipEvent_t start, stop;
	float ms;
};

// ==================================================================
// ==================================================================
using namespace std;

/* BIG = 1/MACHEPF */

#define BIG   16777216.0f
#define MACHEPF 5.9604644775390625E-8f

/* MAXNUMF = 2^128 * (1 - 2^-24) */
#define  MAXNUMF 3.4028234663852885981170418348451692544e38f
/* log(2^-149) */
#define  MINLOGF -103.278929903431851103f		 
#define  PIF 3.141592653589793238f
#define  PIINV  0.318309886183790671538f

/* log( sqrt( 2*pi ) ) */
#define LS2PI  0.91893853320467274178f
#define MAXLGM 2.035093e36
#define MAXLOGF 88.72283905206835f

// sqrt(2pi)
#define s2pi 2.50662827463100050242f

/* log gamma(x+2), -.5 < x < .5 */
__host__ __device__ float polevlfB(float xx)
{
	float t = 6.055172732649237E-004f;
	t = t * xx - 1.311620815545743E-003f;
	t = t * xx + 2.863437556468661E-003f;
	t = t * xx - 7.366775108654962E-003f;
	t = t * xx + 2.058355474821512E-002f;
	t = t * xx - 6.735323259371034E-002f;
	t = t * xx + 3.224669577325661E-001f;
	t = t * xx + 4.227843421859038E-001f;
	return t;
}

/* log gamma(x+1), -.25 < x < .25 */
__host__ __device__ float polevlfC(float xx)
{
	float t = 1.369488127325832E-001f;
	t = t * xx - 1.590086327657347E-001f;
	t = t * xx + 1.692415923504637E-001f;
	t = t * xx - 2.067882815621965E-001f;
	t = t * xx + 2.705806208275915E-001f;
	t = t * xx - 4.006931650563372E-001f;
	t = t * xx + 8.224670749082976E-001f;
	t = t * xx - 5.772156501719101E-001f;
	return t;
}

/* approximation for 0 <= |y - 0.5| <= 3/8 */

__host__ __device__ float polevlfP0(float xx)
{
	float t = -5.99633501014107895267E1f;
	t = t * xx + 9.80010754185999661536E1f;
	t = t * xx - 5.66762857469070293439E1f;
	t = t * xx + 1.39312609387279679503E1f;
	t = t * xx - 1.23916583867381258016E0f;
	return t;
}

__host__ __device__ float p1evlfQ0(float xx)
{
	float t = xx + 1.95448858338141759834E0f;
	t = t * xx + 4.67627912898881538453E0f;
	t = t * xx + 8.63602421390890590575E1f;
	t = t * xx - 2.25462687854119370527E2f;
	t = t * xx + 2.00260212380060660359E2f;
	t = t * xx - 8.20372256168333339912E1f;
	t = t * xx + 1.59056225126211695515E1f;
	t = t * xx - 1.18331621121330003142E0f;
	return t;
}

/* Approximation for interval z = sqrt(-2 log y ) between 2 and 8
* i.e., y between exp(-2) = .135 and exp(-32) = 1.27e-14.
*/

__host__ __device__ float polevlfP1(float xx)
{
	float t = 4.05544892305962419923E0f;
	t = t * xx + 3.15251094599893866154E1f;
	t = t * xx + 5.71628192246421288162E1f;
	t = t * xx + 4.40805073893200834700E1f;
	t = t * xx + 1.46849561928858024014E1f;
	t = t * xx + 2.18663306850790267539E0f;
	t = t * xx - 1.40256079171354495875E-1f;
	t = t * xx - 3.50424626827848203418E-2f;
	t = t * xx - 8.57456785154685413611E-4f;
	return t;
}

__host__ __device__ float p1evlfQ1(float xx)
{
	float t = xx + 1.57799883256466749731E1f;
	t = t * xx + 4.53907635128879210584E1f;
	t = t * xx + 4.13172038254672030440E1f;
	t = t * xx + 1.50425385692907503408E1f;
	t = t * xx + 2.50464946208309415979E0f;
	t = t * xx - 1.42182922854787788574E-1f;
	t = t * xx - 3.80806407691578277194E-2f;
	t = t * xx - 9.33259480895457427372E-4f;
	return t;
}

/* Approximation for interval z = sqrt(-2 log y ) between 8 and 64
* i.e., y between exp(-32) = 1.27e-14 and exp(-2048) = 3.67e-890.
*/

__host__ __device__ float polevlfP2(float xx)
{
	float t = 3.23774891776946035970E0f;
	t = t * xx + 6.91522889068984211695E0f;
	t = t * xx + 3.93881025292474443415E0f;
	t = t * xx + 1.33303460815807542389E0f;
	t = t * xx + 2.01485389549179081538E-1f;
	t = t * xx + 1.23716634817820021358E-2f;
	t = t * xx + 3.01581553508235416007E-4f;
	t = t * xx + 2.65806974686737550832E-6f;
	t = t * xx + 6.23974539184983293730E-9f;
	return t;
}

__host__ __device__ float p1evlfQ2(float xx)
{
	float t = xx + 6.02427039364742014255E0f;
	t = t * xx + 3.67983563856160859403E0f;
	t = t * xx + 1.37702099489081330271E0f;
	t = t * xx + 2.16236993594496635890E-1f;
	t = t * xx + 1.34204006088543189037E-2f;
	t = t * xx + 3.28014464682127739104E-4f;
	t = t * xx + 2.89247864745380683936E-6f;
	t = t * xx + 6.79019408009981274425E-9f;
	return t;
}


__host__ __device__ float lgamf(float xx)
{
	float p, q, w, z, x;
	float nx, tx;
	int i, direction;

	int sgngamf = 1;

	x = xx;
	if (x < 0.0f)
	{
		q = -x;
		w = lgamf(q); /* note this modifies sgngam! */
		p = floorf(q);
		if (p == q)
			goto loverf;
		i = (int)p;
		if ((i & 1) == 0)
			sgngamf = -1;
		else
			sgngamf = 1;
		z = q - p;
		if (z > 0.5f)
		{
			p += 1.0f;
			z = p - q;
		}
		z = q * sinf(PIF * z);
		if (z == 0.0)
			goto loverf;
		z = -logf(PIINV*z) - w;
		return(z);
	}

	if (x < 6.5f)
	{
		direction = 0;
		z = 1.0;
		tx = x;
		nx = 0.0;
		if (x >= 1.5)
		{
			while (tx > 2.5f)
			{
				nx -= 1.0f;
				tx = x + nx;
				z *= tx;
			}
			x += nx - 2.0f;
		iv1r5:
			p = x * polevlfB(x);
			goto cont;
		}
		if (x >= 1.25f)
		{
			z *= x;
			x -= 1.0f; /* x + 1 - 2 */
			direction = 1;
			goto iv1r5;
		}
		if (x >= 0.75f)
		{
			x -= 1.0f;
			p = x * polevlfC(x);
			q = 0.0f;
			goto contz;
		}
		while (tx < 1.5f)
		{
			if (tx == 0.0f)
				goto loverf;
			z *= tx;
			nx += 1.0f;
			tx = x + nx;
		}
		direction = 1;
		x += nx - 2.0f;
		p = x * polevlfB(x);

	cont:
		if (z < 0.0f)
		{
			sgngamf = -1;
			z = -z;
		}
		else
		{
			sgngamf = 1;
		}
		q = logf(z);
		if (direction)
			q = -q;
	contz:
		return(p + q);
	}

	if (x > MAXLGM)
	{
	loverf:
		return(sgngamf * MAXNUMF);	// overflow
	}

	// Note, though an asymptotic formula could be used for x >= 3,
	// there is cancellation error in the following if x < 6.5. 
	q = LS2PI - x;
	q += (x - 0.5f) * logf(x);

	if (x <= 1.0e4)
	{
		z = 1.0f / x;
		p = z * z;
		q += ((6.789774945028216E-004f * p
			- 2.769887652139868E-003f) * p
			+ 8.333316229807355E-002f) * z;
	}
	return(q);
}

//
// Continued fraction expansion #1 for incomplete beta integral.
//
__host__ __device__ float incbcff(float aa, float bb, float xx)
{
	float a, b, x, xk, pk, pkm1, pkm2, qk, qkm1, qkm2;
	float k1, k2, k3, k4, k5, k6, k7, k8;
	float r, t, ans;
	int n;

	a = aa;
	b = bb;
	x = xx;
	k1 = a;
	k2 = a + b;
	k3 = a;
	k4 = a + 1.0f;
	k5 = 1.0f;
	k6 = b - 1.0f;
	k7 = k4;
	k8 = a + 2.0f;

	pkm2 = 0.0f;
	qkm2 = 1.0f;
	pkm1 = 1.0f;
	qkm1 = 1.0f;
	ans = 1.0f;
	r = 0.0f;
	n = 0;
	do
	{

		xk = -(x * k1 * k2) / (k3 * k4);
		pk = pkm1 + pkm2 * xk;
		qk = qkm1 + qkm2 * xk;
		pkm2 = pkm1;
		pkm1 = pk;
		qkm2 = qkm1;
		qkm1 = qk;

		xk = (x * k5 * k6) / (k7 * k8);
		pk = pkm1 + pkm2 * xk;
		qk = qkm1 + qkm2 * xk;
		pkm2 = pkm1;
		pkm1 = pk;
		qkm2 = qkm1;
		qkm1 = qk;

		if (qk != 0)
			r = pk / qk;

		if (r != 0)
		{
			t = fabsf((ans - r) / r);
			ans = r;
		}
		else
			t = 1.0f;

		if (t < MACHEPF) return (ans);

		k1 += 1.0f;
		k2 += 1.0f;
		k3 += 2.0f;
		k4 += 2.0f;
		k5 += 1.0f;
		k6 -= 1.0f;
		k7 += 2.0f;
		k8 += 2.0f;

		if ((fabsf(qk) + fabsf(pk)) > BIG)
		{
			pkm2 *= MACHEPF;
			pkm1 *= MACHEPF;
			qkm2 *= MACHEPF;
			qkm1 *= MACHEPF;
		}
		if ((fabsf(qk) < MACHEPF) || (fabsf(pk) < MACHEPF))
		{
			pkm2 *= BIG;
			pkm1 *= BIG;
			qkm2 *= BIG;
			qkm1 *= BIG;
		}
	} while (++n < 100);

	return(ans);
}

//
// Continued fraction expansion #2 for incomplete beta integral.
//

__host__ __device__ float incbdf(float aa, float bb, float xx)
{
	float a, b, x, xk, pk, pkm1, pkm2, qk, qkm1, qkm2;
	float k1, k2, k3, k4, k5, k6, k7, k8;
	float r, t, ans, z;
	int n;

	a = aa;
	b = bb;
	x = xx;
	k1 = a;
	k2 = b - 1.0f;
	k3 = a;
	k4 = a + 1.0f;
	k5 = 1.0f;
	k6 = a + b;
	k7 = a + 1.0f;
	k8 = a + 2.0f;

	pkm2 = 0.0f;
	qkm2 = 1.0f;
	pkm1 = 1.0f;
	qkm1 = 1.0f;
	z = x / (1.0f - x);
	ans = 1.0f;
	r = 0.0f;
	n = 0;
	do
	{

		xk = -(z * k1 * k2) / (k3 * k4);
		pk = pkm1 + pkm2 * xk;
		qk = qkm1 + qkm2 * xk;
		pkm2 = pkm1;
		pkm1 = pk;
		qkm2 = qkm1;
		qkm1 = qk;

		xk = (z * k5 * k6) / (k7 * k8);
		pk = pkm1 + pkm2 * xk;
		qk = qkm1 + qkm2 * xk;
		pkm2 = pkm1;
		pkm1 = pk;
		qkm2 = qkm1;
		qkm1 = qk;

		if (qk != 0)
			r = pk / qk;
		if (r != 0)
		{
			t = fabsf((ans - r) / r);
			ans = r;
		}
		else
			t = 1.0f;

		if (t < MACHEPF) return ans;	// underflow

		k1 += 1.0f;
		k2 -= 1.0f;
		k3 += 2.0f;
		k4 += 2.0f;
		k5 += 1.0f;
		k6 += 1.0f;
		k7 += 2.0f;
		k8 += 2.0f;

		if ((fabsf(qk) + fabsf(pk)) > BIG)
		{
			pkm2 *= MACHEPF;
			pkm1 *= MACHEPF;
			qkm2 *= MACHEPF;
			qkm1 *= MACHEPF;
		}
		if ((fabsf(qk) < MACHEPF) || (fabsf(pk) < MACHEPF))
		{
			pkm2 *= BIG;
			pkm1 *= BIG;
			qkm2 *= BIG;
			qkm1 *= BIG;
		}
	} while (++n < 100);

	return(ans);
}

__host__ __device__ float incbpsf(float aa, float bb, float xx)
{
	float a, b, x, t, u, y, s;

	a = aa;
	b = bb;
	x = xx;

	y = a * logf(x) + (b - 1.0f)*logf(1.0f - x) - logf(a);
	y -= lgamf(a) + lgamf(b);
	y += lgamf(a + b);


	t = x / (1.0f - x);
	s = 0.0f;
	u = 1.0f;
	do
	{
		b -= 1.0f;
		if (b == 0.0f)
			break;

		a += 1.0f;
		u *= t*b / a;
		s += u;
	} while (fabsf(u) > MACHEPF);

	if (y < MINLOGF)
	{
		s = 0.0f;	// underflow
	}
	else
		s = expf(y) * (1.0f + s);

	return(s);
}


__host__ __device__  float incbetf(float aa, float bb, float xx)
{
	float ans, a, b, t, x, onemx;
	int flag;

	if ((xx <= 0.0f) || (xx >= 1.0f))
	{
		if (xx == 0.0f)
			return(0.0f);
		if (xx == 1.0f)
			return(1.0f);
		return(0.0f);
	}

	onemx = 1.0f - xx;


	// Transformation for small aa.

	if (aa <= 1.0f)
	{
		ans = incbetf(aa + 1.0f, bb, xx);
		t = aa*logf(xx) + bb*logf(1.0f - xx)
			+ lgamf(aa + bb) - lgamf(aa + 1.0f) - lgamf(bb);

		if (t > MINLOGF)
			ans += expf(t);

		return(ans);
	}


	// see if x is greater than the mean.

	if (xx > (aa / (aa + bb)))
	{
		flag = 1;
		a = bb;
		b = aa;
		t = xx;
		x = onemx;
	}
	else
	{
		flag = 0;
		a = aa;
		b = bb;
		t = onemx;
		x = xx;
	}


	// Choose expansion for optimal convergence.

	if (b > 10.0f)
	{
		if (fabsf(b*x / a) < 0.3f)
		{
			t = incbpsf(a, b, x);
			goto bdone;
		}
	}

	ans = x * (a + b - 2.0f) / (a - 1.0f);
	if (ans < 1.0f)
	{
		ans = incbcff(a, b, x);
		t = b * logf(t);
	}
	else
	{
		ans = incbdf(a, b, x);
		t = (b - 1.0f) * logf(t);
	}

	t += a*logf(x) + lgamf(a + b) - lgamf(a) - lgamf(b);
	t += logf(ans / a);

	if (t < MINLOGF)
	{
		t = 0.0f;	// underflow
	}
	else
	{
		t = expf(t);
	}

bdone:
	if (flag)
		t = 1.0f - t;

	return(t);
}

__host__ __device__ float fdtrcf(float a, float b, float x)
{
	float w;

	if ((a < 1.0f) || (b < 1.0f) || (x < 0.0f))
	{
		return(0.0f);
	}

	w = b / (b + a * x);

	return incbetf(0.5f*b, 0.5f*a, w);
}


__host__ __device__ float ndtrif(float yy0)
{
	float y0, x, y, z, y2, x0, x1;
	int code;

	y0 = yy0;
	if (y0 <= 0.0f)
	{
//		mtherr("ndtrif", DOMAIN);
		return(-MAXNUMF);
	}
	if (y0 >= 1.0f)
	{
//		mtherr("ndtrif", DOMAIN);
		return(MAXNUMF);
	}
	code = 1;
	y = y0;
	if (y > (1.0 - 0.13533528323661269189f)) /* 0.135... = exp(-2) */
	{
		y = 1.0f - y;
		code = 0;
	}

	if (y > 0.13533528323661269189f)
	{
		y = y - 0.5f;
		y2 = y * y;
//		x = y + y * (y2 * polevlf(y2, P0, 4) / p1evlf(y2, Q0, 8));
		x = y + y * (y2 * polevlfP0(y2) / p1evlfQ0(y2));
		x = x * s2pi;
		return(x);
	}

	x = sqrtf(-2.0f * logf(y));
	x0 = x - logf(x) / x;

	z = 1.0f / x;
	if (x < 8.0f) /* y > exp(-32) = 1.2664165549e-14 */
//		x1 = z * polevlf(z, P1, 8) / p1evlf(z, Q1, 8);
		x1 = z * polevlfP1(z) / p1evlfQ1(z);
	else
//		x1 = z * polevlf(z, P2, 8) / p1evlf(z, Q2, 8);
		x1 = z * polevlfP2(z) / p1evlfQ2(z);

	x = x0 - x1;
	if (code != 0)
		x = -x;
	return(x);
}

__host__ __device__ float igamcf(float aa, float xx);

__host__ __device__ float igamf(float aa, float xx)
{
	float a, x, ans, ax, c, r;

	a = aa;
	x = xx;
	if ((x <= 0) || (a <= 0))
		return(0.0f);

	if ((x > 1.0f) && (x > a))
		return(1.0f - igamcf(a, x));

	/* Compute  x**a * exp(-x) / gamma(a)  */
	ax = a * logf(x) - x - lgamf(a);
	if (ax < -MAXLOGF)
	{
		//		mtherr("igamf", UNDERFLOW);
		return(0.0f);
	}
	ax = expf(ax);

	/* power series */
	r = a;
	c = 1.0f;
	ans = 1.0f;

	do
	{
		r += 1.0f;
		c *= x / r;
		ans += c;
	} while (c / ans > MACHEPF);

	return(ans * ax / a);
}

__host__ __device__ float igamcf(float aa, float xx)
{
	float a, x, ans, c, yc, ax, y, z;
	float pk, pkm1, pkm2, qk, qkm1, qkm2;
	float r, t;
//	static float big = BIG;

	a = aa;
	x = xx;
	if ((x <= 0) || (a <= 0))
		return(1.0f);

	if ((x < 1.0f) || (x < a))
		return(1.0f - igamf(a, x));

	ax = a * logf(x) - x - lgamf(a);
	if (ax < -MAXLOGF)
	{
//		mtherr("igamcf", UNDERFLOW);
		return(0.0f);
	}
	ax = expf(ax);

	/* continued fraction */
	y = 1.0f - a;
	z = x + y + 1.0f;
	c = 0.0;
	pkm2 = 1.0f;
	qkm2 = x;
	pkm1 = x + 1.0f;
	qkm1 = z * x;
	ans = pkm1 / qkm1;

	do
	{
		c += 1.0f;
		y += 1.0f;
		z += 2.0f;
		yc = y * c;
		pk = pkm1 * z - pkm2 * yc;
		qk = qkm1 * z - qkm2 * yc;
		if (qk != 0)
		{
			r = pk / qk;
			t = fabsf((ans - r) / r);
			ans = r;
		}
		else
			t = 1.0f;
		pkm2 = pkm1;
		pkm1 = pk;
		qkm2 = qkm1;
		qkm1 = qk;
		if (fabsf(pk) > BIG)
		{
			pkm2 *= MACHEPF;
			pkm1 *= MACHEPF;
			qkm2 *= MACHEPF;
			qkm1 *= MACHEPF;
		}
	} while (t > MACHEPF);

	return(ans * ax);
}

__host__ __device__ float igamif(float aa, float yy0)
{
	float a, y0, d, y, x0, lgm;
	int i;

	if (yy0 > 0.5f)
	{
		//mtherr("igamif", PLOSS);
		return 0.0f;
	}

	a = aa;
	y0 = yy0;

	// approximation to inverse function 
	d = 1.0f / (9.0f*a);
	y = (1.0f - d - ndtrif(y0) * sqrtf(d));
	x0 = a * y * y * y;

	lgm = lgamf(a);

	for (i = 0; i<10; i++)
	{
		if (x0 <= 0.0f)
		{
			// mtherr("igamif", UNDERFLOW);
			return(0.0f);
		}
		y = igamcf(a, x0);
		/* compute the derivative of the function at this point */

		d = (a - 1.0f) * logf(x0) - x0 - lgm;
		if (d < -MAXLOGF)
		{
			// mtherr("igamif", UNDERFLOW);
			goto done;
		}
		d = -expf(d);
		/* compute the step to the next approximation of x */
		if (d == 0.0)
			goto done;

		d = (y - y0) / d;
		x0 = x0 - d;
		if (i < 3)
			continue;

		if (fabsf(d / x0) < (2.0f * MACHEPF))
			goto done;
	}

done:
	return(x0);
}


__host__ __device__ float chdtrcf(float dff, float xx)
{
	float df, x;

	df = dff;
	x = xx;

	if ((x < 0.0f) || (df < 1.0f))
	{
//		mtherr("chdtrcf", DOMAIN);
		return(0.0f);
	}
	return(igamcf(0.5f*df, 0.5f*x));
}


__host__ __device__ float chdtrf(float dff, float xx)
{
	float df, x;

	df = dff;
	x = xx;
	if ((x < 0.0f) || (df < 1.0f))
	{
//		mtherr("chdtrf", DOMAIN);
		return(0.0);
	}
	return(igamf(0.5f*df, 0.5f*x));
}


__host__ __device__ float chdtrif(float dff, float yy)
{
	float y, df, x;

	y = yy;
	df = dff;
	if ((y < 0.0f) || (y > 1.0f) || (df < 1.0f))
	{
		return(0.0f);
	}

	x = igamif(0.5f * df, y);
	return(2.0f * x);
}

//
// Return the area under the F-Distribution from x to +infinity.
// This represents the p-value.
//
__host__ __device__ float CumulativeFDistributionComplimentary(float dof1, float dof2, float x)
{
	return fdtrcf(dof1, dof2, x);
}

// ==================================================================
//  Anova kernel
// ==================================================================


template <int BLOCK_SIZEW, int BLOCK_SIZEH, int BLOCK_MASKW> __global__ void
AnovaKernel(
	float * __restrict__ pVoxelSubject,
	int	  * __restrict__ pSNPSubject,
	float * __restrict__ pVoxelSNP,
	int    NumberOfSNPS,
	int    NumberOfSubjects,
	int    NumberOfVoxels)
{
	// Block index
	const int ddy = gridDim.y;

	const int bx = blockIdx.x;
	      int by = blockIdx.y;

	// Thread index
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	//
	// First phase .. compute the between group variance.
	//
	while (by < (NumberOfVoxels+blockDim.y-1)/blockDim.y)
	{

		// Index of the first sub-matrix of VoxelSubject processed by the block
		const int aBegin = NumberOfSubjects * BLOCK_SIZEW * by;

		// Index of the last sub-matrix of VoxelSubject processed by the block
		const int aEnd = aBegin + NumberOfSubjects - 1;

		// Step size used to iterate through the sub-matrices of VoxelSubject
		const int aStep = BLOCK_SIZEW;

		// Index of the first sub-matrix of SNPSubject processed by the block
		const int bBegin = NumberOfSubjects * BLOCK_SIZEH * bx;

		// Step size used to iterate through the sub-matrices of SNPSubject
		const int bStep = BLOCK_SIZEH;

		__shared__ float s_VoxelSubject[BLOCK_SIZEH][BLOCK_SIZEW];
		__shared__ int s_SNPSubject[BLOCK_SIZEH][BLOCK_SIZEW];


		float n0 = 0.0f;
		float n1 = 0.0f;
		float n2 = 0.0f;

		float sum0 = 0.0f;
		float sum1 = 0.0f;
		float sum2 = 0.0f;


		for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
		{
			const int condVox = (a - aBegin + tx < NumberOfSubjects) && (by * blockDim.y + ty < NumberOfVoxels);
			const int condSNP = (b - bBegin + tx < NumberOfSubjects) && (bx * blockDim.x + ty < NumberOfSNPS);

			s_VoxelSubject[ty][tx] = (condVox == 1) ? pVoxelSubject[a + NumberOfSubjects * ty + tx] : 0.0f;
			s_SNPSubject[ty][tx] = (condSNP == 1) ? pSNPSubject[b + NumberOfSubjects * ty + tx] : -1;

			__syncthreads();		// Synchronize to make sure the matrices are loaded

#pragma unroll
			for (int k = 0; k < BLOCK_SIZEW; k++)
			{
				//
				// This access pattern guarantees no shared memory conflicts.
				//
				int SS = s_SNPSubject[tx][(k + tx) & BLOCK_MASKW];
				float VS = s_VoxelSubject[ty][(k + tx) & BLOCK_MASKW];

				const float C0 = (SS == 0);	// Save the predicate result to avoid IF stmts
				const float C1 = (SS == 1);
				const float C2 = (SS == 2);

				n0 += C0;
				n1 += C1;
				n2 += C2;

				sum0 += C0*VS;
				sum1 += C1*VS;
				sum2 += C2*VS;
			}


			// Synchronize to make sure that the preceding
			// computation is done before loading two new
			// sub-matrices of VoxelSubject and SNPSubject in the next iteration.
			__syncthreads();
		}


		const float n = n0 + n1 + n2;
		const float sum = sum0 + sum1 + sum2;

		const float	mean = sum / n;
		const float	mean0 = sum0 / fmaxf(n0, 1.0f);
		const float	mean1 = sum1 / fmaxf(n1, 1.0f);
		const float	mean2 = sum2 / fmaxf(n2, 1.0f);

		const float T0 = mean0 - mean;
		const float T1 = mean1 - mean;
		const float T2 = mean2 - mean;

		const float bg_var = (n0*T0*T0 + n1*T1*T1 + n2*T2*T2) / 2.0f;


		//
		// Second phase, compute the withing group variance.
		//

		float sumsq0 = 0.0f;
		float sumsq1 = 0.0f;
		float sumsq2 = 0.0f;


		for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
		{
			const int condVox = (a - aBegin + tx < NumberOfSubjects) && (by * blockDim.y + ty < NumberOfVoxels);
			const int condSNP = (b - bBegin + tx < NumberOfSubjects) && (bx * blockDim.x + ty < NumberOfSNPS);

			s_VoxelSubject[ty][tx] = (condVox == 1) ? pVoxelSubject[a + NumberOfSubjects * ty + tx] : 0.0f;
			s_SNPSubject[ty][tx] = (condSNP == 1) ? pSNPSubject[b + NumberOfSubjects * ty + tx] : -1;

			__syncthreads();		// Synchronize to make sure the matrices are loaded

#pragma unroll
			for (int k = 0; k < BLOCK_SIZEW; k++)
			{
				//
				// This access pattern guarantees no shared memory conflicts.
				//
				int SS = s_SNPSubject[tx][(k + tx) & BLOCK_MASKW];
				float VS = s_VoxelSubject[ty][(k + tx) & BLOCK_MASKW];

				float C0 = (SS == 0);	// Save the predicate result to avoid IF stmts
				float C1 = (SS == 1);
				float C2 = (SS == 2);

				sumsq0 += C0*(VS - mean0)*(VS - mean0);
				sumsq1 += C1*(VS - mean1)*(VS - mean1);
				sumsq2 += C2*(VS - mean2)*(VS - mean2);
			}

			// Synchronize to make sure that the preceding
			// computation is done before loading two new
			// sub-matrices of VoxelSubject and SNPSubject in the next iteration.
			__syncthreads();
		}

		const int tidx = blockIdx.x * blockDim.x + tx;
		const int tidy = by * blockDim.y + ty;

		const float wg_var = (sumsq0 + sumsq1 + sumsq2) / (n - 3.0f);
		const int c        = NumberOfSNPS * BLOCK_SIZEH * by + BLOCK_SIZEW * bx;

		if (tidx < NumberOfSNPS && tidy < NumberOfVoxels)
			pVoxelSNP[c + NumberOfSNPS * ty + tx] = CumulativeFDistributionComplimentary(2.0f, n - 3.0f, bg_var / wg_var);

		by += ddy;
	}
}

// ==================================================================
//  CopySubset kernel
// ==================================================================

template <typename T> __global__ void
CopySubsetKernel(
	T * Src,
	int *  SrcList,
	T * Dst,
	int SrcH,
	int SrcW,
	int SrcListLen)
{
	const int y	= blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;	

	if (y >= SrcH) return;

	int srow = -1;

	// Zero'th thread in each warp reads the row number

	if ((threadIdx.x & 0x1F) == 0)
		srow = SrcList[y];

	srow = __shfl(srow, 0);	// all threads in the warp read from laneid 0

	// If you put this IF statment before the __shfl instruction, then the right most warps
	// with less than 32 threads will hang up indefinitely.

	if (x >= SrcW) return;

	const int sidx = srow * SrcW + x;
	const int didx = y * SrcW + x;

	T val	  = Src[sidx];
	Dst[didx] = val;
}

// ==================================================================
//  DumpRam
// ==================================================================

void DumpRam(float *dS1, float *dS2, int off, int cnt)
{
	float *hS1 = new float[cnt];
	float *hS2 = new float[cnt];

	hipMemcpy(hS1, dS1+off, cnt * 4, hipMemcpyDeviceToHost);
	hipMemcpy(hS2, dS2+off, cnt * 4, hipMemcpyDeviceToHost);

	delete[] hS1;
	delete[] hS2;
}

// ==================================================================
//  DoKernelAnova
// ==================================================================

#define BLK_SIZEW 16
#define BLK_SIZEH 16
#define BLK_MASKW 0xF

hipError_t DoKernelAnova(
	float *VoxelSubject,
	int *SNPSubject,
	float *VoxelSNP,

	int NumberOfSNPs,
	int NumberOfSubjects,
	int NumberOfVoxels,

	int *VoxelList,
	int VoxelListCount,
	int *SNPList,
	int SNPListCount)
{

	//
	//  1. GPUMalloc VoxelSubjectTemp of size (VoxelListCount, NumberOfSubjects)
	//  2. Use VoxelList to copy the subset of VoxelSubject into VoxelSubjectTemp
	//
	//  3. GPUMalloc SNPSubjectTemp of size (SNPListCount, NumberOfSubjects)
	//  4. Use SNPList to copy the subset of SNPSubject into SNPSubjectTemp
	//
	//  5. call Anova with VoxelSubjectTemp, SNPSubjectTemp, VoxelListCount, SNPListCount, NumberOfSubjects
	//
	//  6. GPUFree VoxelSubjectTemp
	//  7. GPUFree SNPSubjectTemp

	hipError_t error;

	// Special case ... the host wants to process all of the voxels (could be as much as 8000000) but the
	//                  SNPListCount=1. This result is a VERY tall matrix that is 1 element wide. The results 
	//                  are used to show how this 1 SNP affects every voxel in the 3D brain view.
	//
	if (VoxelListCount == NumberOfVoxels && SNPListCount == 1)
	{
		int *SNPSubjectTemp;

		error = hipMalloc(&SNPSubjectTemp, SNPListCount*NumberOfSubjects * sizeof(int));
		if (error != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed on SNPSubjectTemp (error code %s)!\n", hipGetErrorString(error));
			return error;
		}

		{
			dim3 block = dim3(512, 1, 1);
			dim3 grid = dim3((NumberOfSubjects + block.x - 1) / block.x, SNPListCount, 1);
			CopySubsetKernel<int> << <grid, block >> > (SNPSubject, SNPList, SNPSubjectTemp, NumberOfSNPs, NumberOfSubjects, SNPListCount);
			hipDeviceSynchronize();
			error = hipGetLastError();
			if (error != hipSuccess)
			{
				fprintf(stderr, "CopySubsetKernel failed on SNPSubject (error code %s)!\n", hipGetErrorString(error));
				exit(EXIT_FAILURE);
			}
		}

		{
			dim3 block = dim3(BLK_SIZEW, BLK_SIZEH, 1);
			dim3 grid = dim3((SNPListCount + BLK_SIZEW - 1) / BLK_SIZEW, 16383, 1);

			AnovaKernel<BLK_SIZEW, BLK_SIZEH, BLK_MASKW> << <grid, block >> > (VoxelSubject,
				SNPSubjectTemp,
				VoxelSNP,
				SNPListCount,
				NumberOfSubjects,
				VoxelListCount
				);
		}

		hipDeviceSynchronize();

		error = hipGetLastError();
		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to launch (VoxelListCount == NumberOfVoxels && SNPListCount == 1) (error code %s)!\n", hipGetErrorString(error));
		}
		hipFree(SNPSubjectTemp);
	}
	//
	// This code will be called by the host when the data inside the window needs processed.
	//
	else
	{
		float *VoxelSubjectTemp;
		int *SNPSubjectTemp;

		error = hipMalloc(&VoxelSubjectTemp, VoxelListCount*NumberOfSubjects * sizeof(float));
		if (error != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed on VoxelSubjectTemp (error code %s)!\n", hipGetErrorString(error));
			return error;
		}

		error = hipMalloc(&SNPSubjectTemp, SNPListCount*NumberOfSubjects * sizeof(int));
		if (error != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed on SNPSubjectTemp (error code %s)!\n", hipGetErrorString(error));
			hipFree(VoxelSubjectTemp);
			return error;
		}

		{
			dim3 block = dim3(512, 1, 1);
			dim3 grid = dim3((NumberOfSubjects + block.x - 1) / block.x, VoxelListCount, 1);
			CopySubsetKernel<float> << <grid, block >> > (VoxelSubject, VoxelList, VoxelSubjectTemp, NumberOfVoxels, NumberOfSubjects, VoxelListCount);
			hipDeviceSynchronize();
			error = hipGetLastError();
			if (error != hipSuccess)
			{
				fprintf(stderr, "CopySubsetKernel failed on VoxelSubject (error code %s)!\n", hipGetErrorString(error));
				exit(EXIT_FAILURE);
			}
		}

		{
			dim3 block = dim3(512, 1, 1);
			dim3 grid = dim3((NumberOfSubjects + block.x - 1) / block.x, SNPListCount, 1);
			CopySubsetKernel<int> << <grid, block >> > (SNPSubject, SNPList, SNPSubjectTemp, NumberOfSNPs, NumberOfSubjects, SNPListCount);
			hipDeviceSynchronize();
			error = hipGetLastError();
			if (error != hipSuccess)
			{
				fprintf(stderr, "CopySubsetKernel failed on SNPSubject (error code %s)!\n", hipGetErrorString(error));
				exit(EXIT_FAILURE);
			}
		}


		{
			dim3 block = dim3(BLK_SIZEW, BLK_SIZEH, 1);
			dim3 grid = dim3((SNPListCount + BLK_SIZEW - 1) / BLK_SIZEW, (VoxelListCount + BLK_SIZEH - 1) / BLK_SIZEH, 1);

			AnovaKernel<BLK_SIZEW, BLK_SIZEH, BLK_MASKW> << <grid, block >> > (VoxelSubjectTemp,
				SNPSubjectTemp,
				VoxelSNP,
				SNPListCount,
				NumberOfSubjects,
				VoxelListCount
				);

		}

		hipDeviceSynchronize();

		error = hipGetLastError();
		if (error != hipSuccess)
		{
			fprintf(stderr, "Failed to launch (error code %s)!\n", hipGetErrorString(error));
		}

		hipFree(VoxelSubjectTemp);
		hipFree(SNPSubjectTemp);
	}

	return error;
}

// ==================================================================
//  DoKernelVegasTest
// ==================================================================
#define NumThreadsPerBlock 256

__global__ void GenerateRandomSequenceKernel(float *Dest, int NumberOfValues, hiprandState_t* States)
{
	const int gtid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ hiprandState_t RT[NumThreadsPerBlock];
	
	RT[threadIdx.x] = States[gtid];

	for (int i = gtid; i < NumberOfValues; i += blockDim.x*gridDim.x)
	{
		float T = hiprand_normal(&RT[threadIdx.x]);
		Dest[i] = T;
	}
	States[gtid] = RT[threadIdx.x];
}

__global__ void  InitRandomKernel(unsigned int seed, hiprandState_t* States)
{
	const int gtid = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState_t S;
	hiprand_init(gtid << 8, 0, 0, &S);
	States[gtid] = S;
}

__global__ void GenerateRandomVariatesKernel(float *LDMatrix, int LDMatrixSize, float *N01VariatesBuffer, int N01Offset, int Length, float *ResultBuffer)
{
	const int gtid = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = gtid; i < LDMatrixSize*Length; i += blockDim.x*gridDim.x)
	{
		float T = N01VariatesBuffer[N01Offset*LDMatrixSize*Length + i];
		ResultBuffer[i] = T;
	}
}


class KGene
{
public:
	KGene() :
		m_SnpPos(0),
		m_SnpLen(0),
		m_pLDMatrix(0),
		m_LDMatrixSize(0)
	{};

	~KGene()
	{
	}

	void AddSnpPos(int pos, int len);
	void AddLDMatrix(uint64_t N, uint64_t MatrixPtr);

	int GetSnpPos(void) { return m_SnpPos; }
	int GetSnpLen(void) { return m_SnpLen; }
	float *GetLDMatrixPtr(void) { return m_pLDMatrix;  }
	int GetLDMatrixSize(void) { return m_LDMatrixSize;  }

private:
	int		m_SnpPos;
	int		m_SnpLen;
	float*	m_pLDMatrix;
	int		m_LDMatrixSize;
};

void KGene::AddSnpPos(int pos, int len)
{
	m_SnpPos = pos;
	m_SnpLen = len;
}

void KGene::AddLDMatrix(uint64_t N, uint64_t MatrixPtr)
{
	m_pLDMatrix = (float *)MatrixPtr;
	m_LDMatrixSize = (int)N;

	// Just a sanity check to insure the values
	// are equal. Unfortunately this implies that
	// AddSnpPos() is called before AddLDMatrix().
	//
	assert(m_LDMatrixSize == m_SnpLen);
}

#if 0
// =========================================================================================================================
void ComputeObservedPvalues(CGene* G, int GeneNumber, int VoxelNumber, CArray2D<float> &VoxelSNP, CArray2D<float> &VoxelGeneObs)
{
	double sum = 0;

	for (int i = G->GetSnpPos(); i < G->GetSnpPos() + G->GetSnpLen(); i++)
	{
		const float PValue = VoxelSNP(VoxelNumber, i);
		assert((1.0f - PValue) >= 0 && (1.0f - PValue) < 1);

		// I hate to cast the chi2inv as a float, but we are trying to make this app
		// as fast as possible.

		const float ChiSquare = (float)chi2inv(1.0f - PValue, 1);
		sum += ChiSquare;
	}

	VoxelGeneObs(VoxelNumber, GeneNumber) = (float)sum;
	cout << "\tVoxel " << setw(3) << VoxelNumber << " Gene " << setw(3) << GeneNumber;
	cout << " Snp-Pos " << setw(4) << G->GetSnpPos() << " Snp-Len " << setw(4) << G->GetSnpLen() << " Test Statistic " << sum << endl;
}
#endif

struct InvChiSq_functor : public thrust::unary_function<float, float>
{
	__host__ __device__
		float operator()(float x) const
	{
//		return (float)chi2inv(1.0f - PValue, 1);
		return 1.0f / (x*x + 1.0f);
	}
};

struct sumsq_functor
{
	int R;
	int C;
	float *arr;

	sumsq_functor(int _R, int _C, float *_arr) : R(_R), C(_C), arr(_arr) {};

	__host__ __device__
		float operator()(int myC) {
		float sum = 0;
		for (int i = 0; i < R; i++)
		{
			float T = arr[i*C + myC];
			sum += T * T;
		}
		return sum;
	}
};

struct compare_functor
{
	float Thresh;

	compare_functor(float Threshold) : Thresh(Threshold) {};

	__host__ __device__
		float operator()(float X) 
		{
		return X > Thresh ? 1.0f : 0.0f;
		}
};


hipError_t DoKernelVegasTest(float *dVoxelGeneObserved,
	float *dVoxelGeneSim,
	float *dVoxelSNP,
	int *dSNPPosLenPairs,
	uint64_t *dLDMatrixList,
	int NumberOfVoxels,
	int NumberOfGenes,
	int NumberOfSNPs,
	int NumberOfIterations,
	int NumberOfSNPPositionLength
)
{
	// 1. Create the GeneList
	//
	// 2. Compute the VoxelSNPkey vector
	//
	// 3. Convert each SNP p-value to it's chi square value, then reduce
	//	  them according to the VoxelSNPkey vector, and write the result
	//	  to the VoxelGeneObserved matrix.
	//
	// 4. PerformSimulation
	//		for each G in Genes
	//			RandNumberCount = NumberOfVoxels* SNPS in G * Iteration
	//			Launch Kernel to generate random  N(0,1)'s
	//			for each V in Voxels
	//				P = pointer to next set of random N(0,1)'s
	//				Launch MatMul Kernel( LDMatrix_G and P )
	//				Plus reduce SNP p-values to GeneValue, Compare to Observed, Count
	//				Write Count/Iterations to VoxelGeneSim
	//			next V
	//		next G
	//
	// 5. Free GeneList
	//
	int numSMs;
	const int devId = 0;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);


	//
	// STEP 1 -- Create the Gene List.
	//
	vector<KGene*> GeneList;

	int *hSNPPosLenPairs = new int[NumberOfGenes * 2];
	uint64_t *hLDMatrixList = new uint64_t[NumberOfGenes * 2];

	hipError_t err = hipMemcpy(hSNPPosLenPairs, dSNPPosLenPairs, NumberOfGenes * 2 * sizeof(int), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);
	err = hipMemcpy(hLDMatrixList, dLDMatrixList, NumberOfGenes * 2 * sizeof(uint64_t), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);

	int MaxSnpLen = 0;
	for (int i = 0; i < NumberOfGenes; i++)
	{
		KGene* G = new KGene;

		G->AddSnpPos(hSNPPosLenPairs[2 * i], hSNPPosLenPairs[2 * i + 1]);
		G->AddLDMatrix(hLDMatrixList[2 * i], hLDMatrixList[2 * i + 1]);

		if (G->GetSnpLen() > MaxSnpLen)
			MaxSnpLen = G->GetSnpLen();

		GeneList.push_back(G);
	}
	delete[] hSNPPosLenPairs;
	delete[] hLDMatrixList;

	// 
	// STEP 2
	//
	// First build a "mask" that will be used on the GPU to perform a
	//		"reduce by key" with "transformation". The Thrust library
	//		calls this mask a "key".
	//
	unsigned char *dVoxelSNPKey;
	err = hipMalloc((void**)&dVoxelSNPKey, NumberOfVoxels*NumberOfSNPs);
	assert(err == hipSuccess);

	{
		unsigned char *hVoxelSNPKey = new unsigned char[NumberOfVoxels*NumberOfSNPs];
		int SnpSum = 0;
		for (int i = 0; i < GeneList.size(); i++)
		{
			memset(hVoxelSNPKey + SnpSum, i, GeneList[i]->GetSnpLen());
			SnpSum += GeneList[i]->GetSnpLen();
		}
		for (int i = 1; i < NumberOfVoxels; i++)
			memcpy(hVoxelSNPKey + SnpSum*i, hVoxelSNPKey, SnpSum);
		err = hipMemcpy(dVoxelSNPKey, hVoxelSNPKey, SnpSum*NumberOfVoxels, hipMemcpyHostToDevice);
		delete[] hVoxelSNPKey;
	}

	//
	// STEP 3
	//
	// Convert each SNP p-value to it's chi square value, then reduce
	//	  them according to the VoxelSNPkey vector, and write the result
	//	  to the VoxelGeneObserved matrix.

	thrust::device_vector<unsigned char> d_OutputKeys(NumberOfVoxels*NumberOfGenes);

	thrust::pair<thrust::device_vector<unsigned char>::iterator, float * > new_end;
	new_end = thrust::reduce_by_key(thrust::device,
		dVoxelSNPKey,
		dVoxelSNPKey+ NumberOfVoxels*NumberOfSNPs,
		thrust::make_transform_iterator(dVoxelSNP, InvChiSq_functor()),
		d_OutputKeys.begin(),
		dVoxelGeneObserved);

	assert(new_end.first - d_OutputKeys.begin() == NumberOfVoxels*NumberOfGenes);

	//
	// This is kind of gross. Unfortunately I have to copy the dVoxelGeneObserved matrix
	// back to the host because I have to pass each value into the loops below to act
	// as a threshold against the computed Gene p-value.
	//
	float *hVoxelGeneObserved = new float[NumberOfVoxels*NumberOfGenes];
	err = hipMemcpy(hVoxelGeneObserved, dVoxelGeneObserved, NumberOfVoxels*NumberOfGenes * sizeof(float), hipMemcpyDeviceToHost);
	assert(err == hipSuccess);

	//
	// STEP 4
	//
	const int NumBlock = 512;
	const int NumThreads = NumThreadsPerBlock;

	hiprandState_t* States;
	err = hipMalloc((void**)&States, NumBlock * NumThreads * sizeof(hiprandState_t));
	assert(err == hipSuccess);
	InitRandomKernel << <NumBlock, NumThreads >> > (time(0), States);

	float *dRandomNumberBuffer;
	err = hipMalloc(&dRandomNumberBuffer, MaxSnpLen * NumberOfVoxels * NumberOfIterations * sizeof(float));
	assert(err == hipSuccess);

	float *dIterationResultBuffer;
	err = hipMalloc(&dIterationResultBuffer, MaxSnpLen *  NumberOfIterations * sizeof(float));
	assert(err == hipSuccess);

	thrust::device_vector<int> keys(NumberOfIterations, 0);

	//
	// Now loop over every gene ...
	//     and loop over every voxel ...
	//     and compute the VoxelGenSim value.
	//
	for (int g = 0; g < GeneList.size(); g++)
	{
		const int RandomNumberCount = GeneList[g]->GetSnpLen() * NumberOfVoxels * NumberOfIterations;

		GenerateRandomSequenceKernel << <numSMs * 32, NumThreadsPerBlock >> > (dRandomNumberBuffer, RandomNumberCount, States);

		for (int v = 0; v < NumberOfVoxels; v++)
		{
			const int StartingIndex = (v * GeneList[g]->GetSnpLen() *  NumberOfIterations) & (~0x01F);
			GenerateRandomVariatesKernel << <NumBlock, NumThreads >> > (GeneList[g]->GetLDMatrixPtr(), GeneList[g]->GetLDMatrixSize(), dRandomNumberBuffer, v, NumberOfIterations, dIterationResultBuffer);

			//
			// Square every element in  dIterationResultBuffer, Plus-reduce every column, threshold against dVoxelGeneObserved(g,v), Count results.
			// Write Count / Iterations to dVoxelGeneSim(G,Vox)

			thrust::device_vector<float> col_sums(NumberOfIterations);
			thrust::sequence(col_sums.begin(), col_sums.end()); 
			thrust::transform(col_sums.begin(), col_sums.end(), col_sums.begin(), sumsq_functor(GeneList[g]->GetLDMatrixSize(), NumberOfIterations, thrust::raw_pointer_cast(dIterationResultBuffer)));

			thrust::transform(col_sums.begin(), col_sums.end(), col_sums.begin(), compare_functor(hVoxelGeneObserved[v*NumberOfGenes + g]));

			thrust::reduce_by_key(thrust::device, 
				keys.begin(), 
				keys.end(), 
				col_sums.begin(), 
				thrust::make_discard_iterator(), 
				dVoxelGeneSim + v*NumberOfGenes + g );
		}
	}

	thrust::transform(thrust::device, 
		dVoxelGeneSim, 
		dVoxelGeneSim+NumberOfVoxels*NumberOfGenes,
		thrust::make_constant_iterator<float>(NumberOfIterations), 
		dVoxelGeneSim,
		thrust::divides<float>());


	//
	// STEP 5
	//
	delete[] hVoxelGeneObserved;
	hipFree(dIterationResultBuffer);
	hipFree(dRandomNumberBuffer);
	hipFree(States);
	hipFree(dVoxelSNPKey);

	while (!GeneList.empty())
	{
		KGene *G = GeneList.back();
		GeneList.pop_back();
		delete G;
	}

	return hipSuccess;
}





